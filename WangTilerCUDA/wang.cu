#include "hip/hip_runtime.h"
#include <memory>
#include <iostream>
#include <chrono>

#include "hip/hip_runtime.h"
#include ""

#include "noita_random.h"
#include "stb_hbwang.h"
#include "coalhax.h"
#include "spells.h"
#include "wand_levels.h"

//TODO fiddle with these to maximize performance, not sure what the correct configuration is
#define NUMBLOCKS 512
#define BLOCKSIZE 64

typedef unsigned char byte;

#define COLOR_PURPLE 0x7f007f
#define COLOR_BLACK00 0x000000
#define COLOR_BLACK01 0x010101
#define COLOR_BLACK02 0x020202
#define COLOR_WHITE 0xffffff
#define COLOR_YELLOW 0xffff00
#define COLOR_COFFEE 0xc0ffee

__device__ __constant__ uint map_w = 0;
__device__ __constant__ uint map_h = 0;
__device__ __constant__ int worldX = 0;
__device__ __constant__ int worldY = 0;
__device__ __constant__ bool isCoalMines = 0;
__device__ __constant__ uint worldSeedStart = 0;
__device__ __constant__ uint worldSeedCount = 0;
__device__ __constant__ int pwCount = 0;
__device__ __constant__ byte ngPlus = 0;
__device__ __constant__ uint maxChestContents;
__device__ __constant__ uint maxChestsPerWorld;
__device__ __constant__ byte loggingLevel = 0;
__device__ __constant__ byte biomeIndex = 0;

// pudy248 note: If more generation differences occur, this would be the place to start debugging.
#define BCSize 9
__device__ __constant__ unsigned long blockedColors[BCSize] = {
	0x00ac6e, //load_pixel_scene4_alt
	0x70d79e, //load_gunpowderpool_01
	0x70d79f, //???
	0x70d7a1, //load_gunpowderpool_04
	0x7868ff, //load_gunpowderpool_02
	0xc35700, //load_oiltank
	0xff0080, //load_pixel_scene2
	0xff00ff, //???
	0xff0aff, //load_pixel_scene
}; 

//GPU memory doesn't like integers which aren't aligned to 4-byte boundaries, so we have to use these methods for memory accesses.
//Technically, since the CPU code just casts to int*, hard-coding a little-endian format may cause errors, but we'll deal with that when we get there.
__device__ int readUnalignedInt(byte* ptr) {
	//some of these casts and parentheses may be unnecessary but better safe than sorry.
	return 
		(((int)*(signed char*)(ptr + 3)) << 24) | 
		(*(ptr + 2) << 16) | 
		(*(ptr + 1) << 8) | 
		*(ptr + 0);
}

__device__ void writeUnalignedInt(byte* ptr, int val) {
	*(ptr + 3) = val >> 24;
	*(ptr + 2) = (val >> 16) & 0xff;
	*(ptr + 1) = (val >> 8) & 0xff;
	*(ptr + 0) = val & 0xff;
}

__device__
unsigned long createRGB(const byte r, const byte g, const byte b)
{
	return ((r & 0xff) << 16) + ((g & 0xff) << 8) + (b & 0xff);
}

__device__
unsigned long getPos(const uint w, byte f, const uint x, const uint y)
{
	return w * y * f + f * x;
}

__device__
unsigned long getPixelColor(const byte* map, const uint w, const uint x, const uint y)
{
	unsigned long long pos = getPos(w, 3, x, y);
	byte r = map[pos];
	byte g = map[pos + 1];
	byte b = map[pos + 2];
	return createRGB(r, g, b);
}

__device__
void setPixelColor(byte* map, uint w, uint x, uint y, unsigned long color)
{
	unsigned long long pos = getPos(w, 3, x, y);
	byte r = ((color >> 16) & 0xff);
	byte g = ((color >> 8) & 0xff);
	byte b = ((color)&0xff);
	map[pos] = r;
	map[pos + 1] = g;
	map[pos + 2] = b;
}

__device__
void fill(byte* map,
			int w,
			int x1,
			int x2,
			int y1,
			int y2,
			long color)
{
	for (int x = x1; x <= x2; x++)
	{
		for (int y = y1; y <= y2; y++)
		{
			setPixelColor(map, w, x, y, color);
		}
	}
}

struct intPair {
	int x;
	int y;
};

__device__
void floodFill(byte* map,
	uint width,
	uint height,
	int initialX,
	int initialY,
	unsigned long fromColor,
	unsigned long toColor,
	byte* visited,
	intPair* stack)
{
	int stackPtr = 0;
	if (initialX < 0 || initialX >= width || initialY < 0 || initialY >= height)
	{
		return;
	}

	stack[stackPtr++] = {initialX, initialY};
	visited[getPos(width, 1, initialX, initialY)] = true;

	int filled = 0;

	while (stackPtr != 0)
	{
		auto pos = stack[--stackPtr];
		const int x = pos.x;
		const int y = pos.y;

		setPixelColor(map, width, x, y, toColor);
		filled++;

		auto tryNext = [&map, &width, &height, &visited, &fromColor, &toColor, &stackPtr, &stack](int nx, int ny)
		{
			if (nx < 0 || nx >= width || ny < 0 || ny >= height)
			{
			return;
			}

			unsigned long long p = getPos(width, 1, nx, ny);
			if (visited[p] == 1)
			{
			return;
			}

			unsigned long nc = getPixelColor(map, width, nx, ny);
			if (nc != fromColor || nc == toColor)
			{
			return;
			}

			visited[p] = 1;
			stack[stackPtr++] = { nx, ny };
		};
		tryNext(x - 1, y);
		tryNext(x + 1, y);
		tryNext(x, y - 1);
		tryNext(x, y + 1);
	}
}

__device__
void fillC0ffee(
	byte* map,
	uint world_seed,
	byte* visited,
	intPair* stack)
{
	NollaPrng rng = NollaPrng(0);
	rng.SetRandomFromWorldSeed(world_seed);
	rng.Next();
	for (int y = 0; y < map_h; y++)
	{
		for (int x = 0; x < map_w; x++)
		{
			long c = getPixelColor(map, map_w, x, y);
			if (c != COLOR_COFFEE)
			{
				continue;
			}
			long to = COLOR_BLACK00;
			double f = rng.Next();
			if (f <= 0.5) // BIOME_RANDOM_BLOCK_CHANCE
			{
				to = COLOR_WHITE;
			}
			floodFill(map, map_w, map_h, x, y, COLOR_COFFEE, to, visited, stack);
		}
	}
}

__device__
NollaPrng GetRNG(int map_w, uint world_seed)
{
	NollaPrng rng = NollaPrng();
	rng.SetRandomFromWorldSeed(world_seed);
	rng.Next();
	int length = (int)((unsigned long long)((long long)map_w * -0x2e8ba2e9) >> 0x20);
	int iters = ((length >> 1) - (length >> 0x1f)) * 0xb + ((uint)world_seed / 0xc) * -0xc + world_seed + map_w;
	if (0 < iters)
	{
		do
		{
			rng.Next();
			iters -= 1;
		} while (iters != 0);
	}
	return rng;
}

__device__
void doCoalMineHax(
	byte* map,
	int width,
	int height)
{
	for (int y = 0; y < height; y++)
	{
		for (int x = 0; x < width; x++)
		{
			long o = getPos(256, 3, x, y);
			long i = getPos(width, 3, x, y);
			long pix = createRGB(coalmine_overlay[o], coalmine_overlay[o + 1], coalmine_overlay[o + 2]);
			if (pix == 0x4000)
			{ // green. Looks like air?
				//pudy248 note: is not actually air, this is the main rock portion of the overlay
				map[i] = 0xFF;
				map[i + 1] = 0xFF;
				map[i + 2] = 0xFF;
			}
			if (pix == 0x0040)
			{ // blue. Looks like air?
				map[i] = 0x00;
				map[i + 1] = 0x00;
				map[i + 2] = 0x00;
			}
			if (pix == 0xFEFEFE)
			{ // white. Stairs. rock_static_intro
				// But in the debug it's not shown?
				// map[i] = 0x0a;
				// map[i + 1] = 0x33;
				// map[i + 2] = 0x44;
				map[i] = 0xFF;
				map[i + 1] = 0xFF;
				map[i + 2] = 0xFF;
			}
			__syncthreads();
		}
	}
}


__device__
bool contains(const unsigned long arr[BCSize], unsigned long val)
{ 
	for (int i = 0; i < BCSize; i++)
		if (arr[i] == val) return true;
	return false;
};

__device__
void blockOutRooms(
	byte* map,
	int width,
	int height)
{
	int increment = dTileSet.short_side_len;
	for (int y = 1; y < height - 1; y+=increment)
	{
		for (int x = 1; x < width; x+=increment)
		{
			long color = getPixelColor(map, width, x, y);
			if (!contains(blockedColors, color))
			{
			continue;
			}
			int startX = x + 1;
			int endX = x + 1;
			int startY = y + 1;
			int endY = y + 1;
			bool foundEnd = false;
			while (!foundEnd && endX < width)
			{
			long c = getPixelColor(map, width, endX, startY);
			if (c == COLOR_BLACK00)
			{
				endX += 1;
				continue;
			};
			endX -= 1;
			foundEnd = true;
			}
			if (endX >= width)
			{
			endX = width - 1;
			}
			foundEnd = false;
			while (!foundEnd && endY < height)
			{
			long c = getPixelColor(map, width, startX, endY);
			if (c == COLOR_BLACK00)
			{
				endY += 1;
				continue;
			};
			endY -= 1;
			foundEnd = true;
			}
			if (endY >= height)
			{
			endY = height - 1;
			}
			fill(map, width, startX, endX, startY, endY, COLOR_WHITE);
		}
	}
}

__device__
const int BIOME_PATH_FIND_WORLD_POS_MIN_X = 159;
__device__
const int BIOME_PATH_FIND_WORLD_POS_MAX_X = 223;
__device__
const int WORLD_OFFSET_X = 35;

//__device__ __shared__ intPair stackCache[BLOCKSIZE * 4];
//__device__ __shared__ byte stackSize[BLOCKSIZE];

class Search
{
public:
	byte* map;

	byte* visited;
	intPair* queueMem;

	byte pathFound;
	int queueSize;
	int targetX;
	int targetY;
	int threadIdx;

	intPair stackCache[4];
	byte stackSize;

	__device__
	void findPath(int x, int y)
	{
		int rmw = map_w; //register map width
		int rmh = map_h; //register map height
		while (queueSize > 0 && pathFound != 1)
		{
			stackSize = 0;
			intPair n = Pop();
			//if((n.x + n.y) % 2 == 0) setPixelColor(map, register_mapW, n.x, n.y, COLOR_PURPLE);
			if (n.x != -1) {
				if (atTarget(n))
				{
					pathFound = 1;
				}
				tryNext(n.x, n.y + 1, rmw, rmh);
				tryNext(n.x - 1, n.y, rmw, rmh);
				tryNext(n.x + 1, n.y, rmw, rmh);
				tryNext(n.x, n.y - 1, rmw, rmh);
			}
			Push();
		}
	}

	__device__ intPair Pop() {
		return queueMem[--queueSize];
	}
	
	__device__ void Push() {
		while (stackSize > 0) {
			queueMem[queueSize++] = stackCache[--stackSize];
		}
	}

	__device__
	void tryNext(int x, int y, int rmw, int rmh)
	{
		if (x >= 0 && y >= 0 && x < rmw && y < rmh) {
			if (visited[y * rmw + x] == 0 && traversable(x, y, rmw))
			{
				visited[y * rmw + x] = 1;
				stackCache[stackSize++] = { x, y };
			}
		}
	}

	__device__ bool traversable(int x, int y, int rmw)
	{
		long c = getPixelColor(map, rmw, x, y);

		return c == COLOR_BLACK00 || c == COLOR_COFFEE;
	}
	__device__ bool atTarget(intPair n)
	{
		return targetY == n.y;
	}
	__device__ int Manhattan(int x, int y)
	{
		int dx = abs(x - targetX);
		int dy = abs(y - targetY);
		return (dx + dy);
	}
	__device__ int ManhattanDown(int x, int y)
	{
		int dy = abs(y - targetY);
		return (dy);
	}
};

__device__ __shared__ Search dSearch[BLOCKSIZE];

__device__
bool isMainPath()
{
	int fill_x_from = (BIOME_PATH_FIND_WORLD_POS_MIN_X - (worldX - WORLD_OFFSET_X) * 512.0) / 10;
	int fill_x_to = fill_x_from + (BIOME_PATH_FIND_WORLD_POS_MAX_X - BIOME_PATH_FIND_WORLD_POS_MIN_X) / 10;
	return fill_x_to > 0 && fill_x_from > 0 && map_w > fill_x_from && fill_x_to < map_w + fill_x_from;
}

__device__
int fillMainPath(
	byte* map)
{
	int fill_x_from = (BIOME_PATH_FIND_WORLD_POS_MIN_X - (worldX - WORLD_OFFSET_X) * 512.0) / 10;
	int fill_x_to = fill_x_from + (BIOME_PATH_FIND_WORLD_POS_MAX_X - BIOME_PATH_FIND_WORLD_POS_MIN_X) / 10;
	fill(map, map_w, fill_x_from, fill_x_to, 0, 6, COLOR_BLACK00);
	return fill_x_from;
}


static const char* _cudaGetErrorEnum(hipError_t error) {
	return hipGetErrorName(error);
}

template <typename T>
void check(T result, char const* const func, const char* const file,
	int const line) {
	if (result) {
		fprintf(stderr, "CUDA error at %s:%d code=%d(%s) \"%s\" \n", file, line,
			static_cast<unsigned int>(result), _cudaGetErrorEnum(result), func);
		exit(EXIT_FAILURE);
	}
}

// This will output the proper CUDA error strings in the event
// that a CUDA host call returns an error
#define checkCudaErrors(val) check((val), #val, __FILE__, __LINE__)

// This will output the proper error string when calling hipGetLastError
#define getLastCudaError(msg) __getLastCudaError(msg, __FILE__, __LINE__)

inline void __getLastCudaError(const char* errorMessage, const char* file,
	const int line) {
	hipError_t err = hipGetLastError();

	if (hipSuccess != err) {
		fprintf(stderr,
			"%s(%i) : getLastCudaError() CUDA error :"
			" %s : (%d) %s.\n",
			file, line, errorMessage, static_cast<int>(err),
			hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

// This will only print the proper error string when calling hipGetLastError
// but not exit program incase error detected.
#define printLastCudaError(msg) __printLastCudaError(msg, __FILE__, __LINE__)

inline void __printLastCudaError(const char* errorMessage, const char* file,
	const int line) {
	hipError_t err = hipGetLastError();

	if (hipSuccess != err) {
		fprintf(stderr,
			"%s(%i) : getLastCudaError() CUDA error :"
			" %s : (%d) %s.\n",
			file, line, errorMessage, static_cast<int>(err),
			hipGetErrorString(err));
	}
}

__host__ __device__
int GetGlobalPosX(int x, int y, int px, int py)
{
	if (x == 14)
	{
		py -= 10;
	}
	int gx = (int)(((x - 35) * 512) / 10) * 10 + px - 15;
	//int gy = (int)(((y - 14) * 512) / 10) * 10 + py - 3;
	return gx;
}
__host__ __device__
int GetGlobalPosY(int x, int y, int px, int py)
{
	if (y == 14)
	{
		py -= 10;
	}
	//int gx = (int)(((x - 35) * 512) / 10) * 10 + px - 15;
	int gy = (int)(((y - 14) * 512) / 10) * 10 + py - 3;
	return gy;
}

//why in god's name does the game store seed positions as 6 char strings???
__host__ __device__ int roundRNGPos(int num) {
	if (-1000000 < num && num < 1000000) return num;
	else if (-10000000 < num && num < 10000000) return num - (num % 10) + (num % 10 >= 5 ? 10 : 0);
	else if (-100000000 < num && num < 100000000) return num - (num % 100) + (num % 100 >= 50 ? 100 : 0);
	return num;
}

// 0 gold_nuggets
// 1 chest_to_gold
// 2 rain_gold
// 3 bomb
// 4 powder
// 5 potion_normal
// 6 potion_secret
// 7 potion_random_material
// 8 potions_pps
// 9 potions_ssr
// 10 kammi
// 11 kuu
// 12 paha_silma
// 13 chaos_die
// 14 shiny_orb
// 15 ukkoskivi
// 16 kiuaskivi
// 17 vuoksikivi
// 18 kakkakikkare
// 19 runestone_light
// 20 runestone_fire
// 21 runestone_magma
// 22 runestone_weight
// 23 runestone_emptiness
// 24 runestone_edges
// 25 runestone_metal
// 26 random_spell
// 27 spell_refresh
// 28 heart_normal
// 29 heart_mimic
// 30 large_heart
// 31 full_heal
// 32 wand_T1
// 33 wand_T1NS
// 34 wand_T2
// 35 wand_T2NS
// 36 wand_T3
// 37 wand_T3NS
// 38 wand_T4
// 39 wand_T4NS
// 40 wand_T5
// 41 wand_T5NS
// 42 wand_T6
// 43 wand_T6NS
// 
// 44 egg_purple
// 45 egg_slime
// 46 egg_monster
// 47 broken_wand
// 
// 64 wand_T10NS
// 65 wand_T1B
// 66 wand_T2B
// 67 wand_T3B
// 68 wand_T4B
// 69 wand_T5B
// 70 wand_T6B
// 
// random spell: 1xxxxxx0
// xxxxx = # of random calls to make
// 
// 253 sampo
// 255 orb

__device__ int MakeRandomCard(NoitaRandom* random) {
	int res = 0;
	char valid = 0;
	while (valid == 0) {
		int itemno = random->Random(0, 392);
		Spell item = all_spells[itemno];
		double sum = 0;
		for (int i = 0; i < 11; i++) sum += item.spawn_probabilities[i];
		if (sum > 0) {
			valid = 1;
			res = itemno;
		}
	}
	return res;
}

__device__ void CheckNormalChestLoot(int x, int y, uint worldSeed, byte expandSpells, byte* writeLoc)
{
	writeUnalignedInt(writeLoc, x);
	writeUnalignedInt(writeLoc + 4, y);
	byte* contents = writeLoc + 9;
	NoitaRandom random = NoitaRandom(worldSeed);
	random.SetRandomSeed(roundRNGPos(x)+509.7, y+683.1);

	int idx = 0;
	int count = 1;
	while (count > 0)
	{
		if (idx >= maxChestContents) {
			if(loggingLevel >= 3) printf("Chest contents overflow in seed %i!\n", worldSeed);
			break;
		}
		count--;
		int rnd = random.Random(1, 100);
		if (rnd <= 7) contents[idx++] = 3;
		else if (rnd <= 40) 
		{
			rnd = random.Random(0, 100);

			rnd = random.Random(0, 100);
			if (rnd > 99)
			{
				int tamount = random.Random(1, 3);
				for (int i = 0; i < tamount; i++)
				{
					random.Random(-10, 10);
					random.Random(-10, 5);
				}

				if (random.Random(0, 100) > 50)
				{
					tamount = random.Random(1, 3);
					for (int i = 0; i < tamount; i++) {
						random.Random(-10, 10);
						random.Random(-10, 5);
					}
				}
				if (random.Random(0, 100) > 80) {
					tamount = random.Random(1, 3);
					for (int i = 0; i < tamount; i++) {
						random.Random(-10, 10);
						random.Random(-10, 5);
					}
				}
			}
			else {
				random.Random(-10, 10);
				random.Random(-10, 5);
			}
			contents[idx++] = 0;
		}
		else if (rnd <= 50)
		{
			rnd = random.Random(1, 100);
			if (rnd <= 94) contents[idx++] = 5;
			else if (rnd <= 98) contents[idx++] = 4;
			else
			{
				rnd = random.Random(0, 100);
				if (rnd <= 98) contents[idx++] = 6;
				else contents[idx++] = 7;
			}
		}
		else if (rnd <= 54) contents[idx++] = 27;
		else if (rnd <= 60)
		{
			byte opts[8] = { 10, 11, 15, 12, 16, 127, 13, 14 };
			rnd = random.Random(0, 7);
			byte opt = opts[rnd];
			if (opt == 127)
			{
				byte r_opts [7] = {19, 20, 21, 22, 23, 24, 25};
				rnd = random.Random(0, 6);
				byte r_opt = r_opts[rnd];
				contents[idx++] = r_opt;
			}
			else
			{
				contents[idx++] = opt;
			}
		}
		else if (rnd <= 65) 
		{
			int amount = 1;
			int rnd2 = random.Random(0, 100);
			if (rnd2 <= 50) amount = 1;
			else if (rnd2 <= 70) amount += 1;
			else if (rnd2 <= 80) amount += 2;
			else if (rnd2 <= 90) amount += 3;
			else amount += 4;

			for (int i = 0; i < amount; i++) {
				random.Random(0, 1);
				if (expandSpells > 0) {
					int randCTR = random.randomCTR;
					contents[idx++] = (randCTR << 1) | 0x80;
				}
				MakeRandomCard(&random);
			}

			if(expandSpells == 0)
				contents[idx++] = 26;
		}
		else if (rnd <= 84)
		{
			rnd = random.Random(0, 100);
			if (rnd <= 25) contents[idx++] =32;
			else if (rnd <= 50) contents[idx++] = 33;
			else if (rnd <= 75) contents[idx++] = 34;
			else if (rnd <= 90) contents[idx++] = 35;
			else if (rnd <= 96) contents[idx++] = 36;
			else if (rnd <= 98) contents[idx++] = 37;
			else if (rnd <= 99) contents[idx++] = 38;
			else contents[idx++] = 39;
		}
		else if (rnd <= 95)
		{
			rnd = random.Random(0, 100);
			if (rnd <= 88) contents[idx++] = 28;
			else if (rnd <= 89) contents[idx++] = 29;
			else if (rnd <= 99) contents[idx++] = 30;
			else contents[idx++] = 31;
		}
		else if (rnd <= 98) contents[idx++] = 1;
		else if (rnd <= 99) count += 2;
		else count += 3;
	}

	*(writeLoc + 8) = (byte)idx;
}

__device__ void CheckGreatChestLoot(int x, int y, uint worldSeed, byte* writeLoc)
{
	writeUnalignedInt(writeLoc, x);
	writeUnalignedInt(writeLoc + 4, y);
	byte* contents = writeLoc + 9;
	NoitaRandom random = NoitaRandom(worldSeed);
	random.SetRandomSeed(roundRNGPos(x), y);

	int idx = 0;
	int count = 1;

	if (random.Random(0, 100000) >= 100000)
	{
		count = 0;
		if (random.Random(0, 1000) == 999) contents[idx++] = 255;
		else contents[idx++] = 253;
	}

	while (count != 0)
	{
		if (idx >= maxChestContents) {
			if (loggingLevel >= 3) printf("Chest contents overflow in seed %i!\n", worldSeed);
			break;
		}
		count--;
		int rnd = random.Random(1, 100);

		if (rnd <= 30)
		{
			rnd = random.Random(0, 100);
			if (rnd <= 30)
				contents[idx++] = 8;
			else
				contents[idx++] = 9;
		}
		else if (rnd <= 33)
		{
			contents[idx++] = 2;
		}
		else if (rnd <= 38)
		{
			rnd = random.Random(1, 30);
			if (rnd == 30)
			{
				contents[idx++] = 18;
			}
			else contents[idx++] = 17;
		}
		else if (rnd <= 39)
		{
			rnd = random.Random(0, 100);
			if (rnd <= 25) contents[idx++] = 36;
			else if (rnd <= 50) contents[idx++] = 37;
			else if (rnd <= 75) contents[idx++] = 38;
			else if (rnd <= 90) contents[idx++] = 39;
			else if (rnd <= 96) contents[idx++] = 40;
			else if (rnd <= 98) contents[idx++] = 41;
			else if (rnd <= 99) contents[idx++] = 42;
			else contents[idx++] = 43;
		}
		else if (rnd <= 60)
		{
			rnd = random.Random(0, 100);
			if (rnd <= 89) contents[idx++] = 28;
			else if (rnd <= 99) contents[idx++] = 30;
			else contents[idx++] = 31;
		}
		else if (rnd <= 99) count += 2;
		else count += 3;
	}
	*(writeLoc + 8) = (byte)idx;
}

__device__ void CheckItemPedestalLoot(int x, int y, uint worldSeed, byte* writeLoc) 
{
	writeUnalignedInt(writeLoc, x);
	writeUnalignedInt(writeLoc + 4, y);
	*(writeLoc + 8) = 1;
	byte* contents = writeLoc + 9;

	NoitaRandom random = NoitaRandom(worldSeed);
	random.SetRandomSeed(x + 425, y - 243);
	int rnd = random.Random(1, 91);

	if (rnd <= 65)
		contents[0] = 5;
	else if (rnd <= 70)
		contents[0] = 4;
	else if (rnd <= 71)
		contents[0] = 13;
	else if (rnd <= 72) {
		byte r_opts[7] = { 19, 20, 21, 22, 23, 24, 25 };
		rnd = random.Random(0, 6);
		byte r_opt = r_opts[rnd];
		contents[0] = r_opt;
	}
	else if (rnd <= 73)
		contents[0] = 44;
	else if (rnd <= 77)
		contents[0] = 45;
	else if (rnd <= 79)
		contents[0] = 46;
	else if (rnd <= 83)
		contents[0] = 16;
	else if (rnd <= 85)
		contents[0] = 15;
	else if (rnd <= 89)
		contents[0] = 47;
	else
		contents[0] = 14;
}

__device__ void spawnHeart(int x, int y, uint seed, byte greedCurse, byte expandSpells, byte* writeLoc)
{
	NoitaRandom random = NoitaRandom(seed);
	if (loggingLevel >= 5) printf("Spawning heart: %i, %i\n", x, y);
	float r = random.ProceduralRandomf(x, y, 0, 1);
	float heart_spawn_percent = 0.7f;

	if (r <= heart_spawn_percent && r > 0.3)
	{
		random.SetRandomSeed(x + 45, y - 2123);
		int rnd = random.Random(1, 100);
		if (rnd <= 90 || y < 512 * 3)
		{
			rnd = random.Random(1, 1000);

			if (rnd >= 1000)
				CheckGreatChestLoot(x, y, seed, writeLoc);
			else 
				CheckNormalChestLoot(x, y, seed, expandSpells, writeLoc);
		}
	}
}

__device__ void spawnChest(int x, int y, uint seed, byte greedCurse, byte expandSpells, byte* writeLoc)
{
	NoitaRandom random = NoitaRandom(seed);
	if(loggingLevel >= 5) printf("Spawning guaranteed chest: %i, %i\n", x, y);
	random.SetRandomSeed(x, y);
	int super_chest_spawn_rate = greedCurse > 0 ? 100 : 2000;
	int rnd = random.Random(1, super_chest_spawn_rate);

	if (rnd >= super_chest_spawn_rate - 1)
		CheckGreatChestLoot(x, y, seed, writeLoc);
	else
		CheckNormalChestLoot(x, y, seed, expandSpells, writeLoc);
}

__device__ void spawnPotion(int x, int y, uint seed, byte greedCurse, byte expandSpells, byte* writeLoc)
{
	NoitaRandom random = NoitaRandom(seed);
	if (loggingLevel >= 5) printf("Spawning item pedestal: %i, %i\n", x, y);
	float rnd = random.ProceduralRandomf(x, y, 0, 1);

	if (rnd > 0.65f)
	{
		if (loggingLevel >= 5) printf("Spawning item on pedestal: %i, %i\n", x+5, y-4);
		CheckItemPedestalLoot(x + 5, y - 4, seed, writeLoc);
	}
}

__device__ void spawnPixelScene(int x, int y, uint seed, byte oiltank, byte greedCurse, byte expandSpells, byte* writeLoc)
{
	NoitaRandom random = NoitaRandom(seed);
	random.SetRandomSeed(x, y);
	if (loggingLevel >= 5) printf("Spawning pixel scene: %i, %i\n", x, y);
	int rnd = random.Random(1, 100);
	if (rnd <= 50 && oiltank == 0 || rnd > 50 && oiltank > 0) {
		float rnd2 = random.ProceduralRandomf(x, y, 0, 1) * 3;
		if (0.5f < rnd2 && rnd2 < 1) {
			spawnChest(x + 94, y + 224, seed, greedCurse, expandSpells, writeLoc);
		}
	}
}

__device__ void spawnPixelScene1(int x, int y, uint seed, byte greedCurse, byte expandSpells, byte* writeLoc) {
	spawnPixelScene(x, y, seed, 0, greedCurse, expandSpells, writeLoc);
}

__device__ void spawnOilTank(int x, int y, uint seed, byte greedCurse, byte expandSpells, byte* writeLoc) {
	spawnPixelScene(x, y, seed, 1, greedCurse, expandSpells, writeLoc);
}

__device__ void spawnWand(int x, int y, uint seed, byte greedCurse, byte expandSpells, byte* writeLoc) {
	NoitaRandom random = NoitaRandom(seed);
	float r = random.ProceduralRandomf(x, y, 0, 1);
	if (r < 0.47) return;
	r = random.ProceduralRandomf(x - 11.431, y + 10.5257, 0, 1);
	if (r < 0.755) return;

	int nx = x - 5;
	int ny = y - 14;
	BiomeWands wandSet = wandLevels[biomeIndex];
	int sum = 0;
	for (int i = 0; i < wandSet.count; i++) sum += wandSet.levels[i].prob;
	r = random.ProceduralRandomf(nx, ny, 0, 1) * sum;
	for (int i = 0; i < wandSet.count; i++) {
		if (r <= wandSet.levels[i].prob) {
			writeUnalignedInt(writeLoc, nx+5);
			writeUnalignedInt(writeLoc + 4, ny+5);
			*(writeLoc + 8) = 1;
			*(writeLoc + 9) = wandSet.levels[i].id;
			return;
		}
		r -= wandSet.levels[i].prob;
	}
}

__device__ size_t sizeOfChest() {
	return 9 + maxChestContents;
}

__device__ size_t sizeOfChestSegment() {
	return sizeof(uint) + sizeOfChest() * maxChestsPerWorld * (2 * pwCount + 1);
}

__global__
void blockRoomBlock(
	byte* block,
	byte* validBlock,
	bool skipValid)
{
	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	uint stride = blockDim.x * gridDim.x;
	for (int idx = index; idx < worldSeedCount; idx += stride) {
		if (skipValid || !validBlock[idx]) {
			byte* segment = block + idx * (3 * map_w * map_h);
			blockOutRooms(segment, map_w, map_h);
		}
	}
}


__global__
void blockFillC0FFEE(
	uint* seeds,
	byte* block,
	byte* validBlock,
	byte* visitedBlock,
	intPair* stackBlock,
	bool skipValid)
{
	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	uint stride = blockDim.x * gridDim.x;
	for (int idx = index; idx < worldSeedCount; idx += stride) {
		if (skipValid || !validBlock[idx]) {
			byte* segment = block + idx * (3 * map_w * map_h);
			byte* visited = visitedBlock + idx * (map_w * map_h);
			intPair* stack = stackBlock + idx * (map_w + map_h);
			uint worldSeed = seeds[idx];

			memset(visited, 0, map_w * map_h);

			fillC0ffee(segment, worldSeed, visited, stack);
		}
	}
}

__global__
void blockIsValid(
	byte* mapBlock,
	byte* validBlock,
	byte* sVisitedBlock,
	intPair* dQueueMem,
	bool skipValid)
{
	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	uint stride = blockDim.x * gridDim.x;
	bool mainPath = isMainPath();


	for (int idx = index; idx < worldSeedCount; idx += stride) {
		if (skipValid || !validBlock[idx]) {
			byte* mapSegment = mapBlock + idx * 3 * map_w * map_h;
			byte* visitedSegment = sVisitedBlock + idx * map_w * map_h;
			intPair* queueMemSegment = dQueueMem + idx * (map_w + map_h);

			uint path_start_x = 0;
			if (mainPath)
			{
				if (isCoalMines)
				{
					path_start_x = 0x8e;
				}
				else
				{
					path_start_x = fillMainPath(mapSegment);
				}
			}

			int x = path_start_x;

			if (!mainPath) {
				while (x < map_w)
				{
					long c = getPixelColor(mapSegment, map_w, x, 0);
					if (c != COLOR_BLACK00)
					{
						x++;
						continue;
					}
					else break;
				}
			}

			dSearch[threadIdx.x].map = mapSegment;
			dSearch[threadIdx.x].visited = visitedSegment;
			dSearch[threadIdx.x].queueMem = queueMemSegment;
			dSearch[threadIdx.x].queueSize = 1;
			dSearch[threadIdx.x].threadIdx = threadIdx.x;
			dSearch[threadIdx.x].targetX = x;
			dSearch[threadIdx.x].targetY = map_h - 1;
			dSearch[threadIdx.x].pathFound = 0;

			dSearch[threadIdx.x].visited[x] = 1;
			setPixelColor(dSearch[threadIdx.x].map, map_w, x, 0, COLOR_PURPLE);
			queueMemSegment[0] = { x, 0 };

			dSearch[threadIdx.x].stackSize = 0;

			dSearch[threadIdx.x].findPath(path_start_x, 0);

			validBlock[idx] = dSearch[threadIdx.x].pathFound;
		}
	}
}

__global__ 
void blockCoalMineHax(
	byte* block, 
	byte* validBlock,
	bool skipValid)
{
	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	uint stride = blockDim.x * gridDim.x;
	for (int idx = index; idx < worldSeedCount; idx += stride) {
		if (skipValid || !validBlock[idx]) {
			byte* segment = block + (idx * (3 * map_w * map_h));
			doCoalMineHax(segment, map_w, map_h);
		}
	}
}

__global__
void blockInitRNG(
	uint* seeds,
	NollaPrng* rngBlock1,
	NollaPrng* rngBlock2)
{
	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	uint stride = blockDim.x * gridDim.x;
	for (int idx = index; idx < worldSeedCount; idx += stride) {
		uint worldSeed = seeds[idx];
		rngBlock1[idx] = GetRNG(map_w, worldSeed);
		rngBlock2[idx] = NollaPrng(0);
	}
}

__global__
void blockUpdateRNG(
	NollaPrng* rngBlock1,
	NollaPrng* rngBlock2)
{
	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	uint stride = blockDim.x * gridDim.x;
	for (int idx = index; idx < worldSeedCount; idx += stride) {
		rngBlock2[idx].Seed = rngBlock1[idx].Next() * 2147483645.0;
		rngBlock2[idx].Next();
	}
}

__global__
void buildTS(
	byte* data,
	int tiles_w,
	int tiles_h)
{
	stbhw_build_tileset_from_image(data, tiles_w * 3, tiles_w, tiles_h);
}
__global__
void freeTS()
{
	stbhw_free_tileset();
}

__global__
void blockGenerateMap(
	byte* resBlock,
	NollaPrng* rngBlock,
	byte* validBlock,
	bool skipValid)
{
	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	uint stride = blockDim.x * gridDim.x;
	for (int idx = index; idx < worldSeedCount; idx += stride) {
		if (skipValid || !validBlock[idx]) {
			byte* res = resBlock + idx * (3 * map_w * (map_h + 4));

			stbhw_generate_image(res, map_w * 3, map_w, map_h + 4, &StaticRandom, rngBlock + idx);
		}
	}
}

__global__
void blockMemcpyOffset(
	byte* fromBlock,
	byte* toBlock,
	byte* validBlock,
	bool skipValid)
{
	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	uint stride = blockDim.x * gridDim.x;
	for (int idx = index; idx < worldSeedCount; idx += stride) {
		if (skipValid || !validBlock[idx]) {
			byte* from = fromBlock + idx * (3 * map_w * (map_h + 4));
			byte* to = toBlock + idx * (3 * map_w * map_h);

			memcpy(to, from, 3 * map_w * map_h);
		}
	}
}

//prepare your eyes for some of the most horrific pointer code ever created
__global__ void blockCheckSpawnables(
	uint* seeds,
	byte* mapBlock,
	byte* retArray,
	byte* validBlock,
	byte greedCurse,
	byte checkItems,
	byte expandSpells,
	byte checkWands) {
	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	uint stride = blockDim.x * gridDim.x;
	for (int idx = index; idx < worldSeedCount; idx += stride) {
		byte* retSegment = retArray + idx * sizeOfChestSegment();
		memset(retSegment, 0, sizeOfChestSegment());
		if (validBlock[idx]) {
			uint worldSeed = seeds[idx];
			byte* map = mapBlock + idx * (3 * map_w * map_h);
			int chestIdx = 0;
			bool densityExceeded = false;

			void (*spawnFuncs[6])(int, int, uint, byte, byte, byte*) = { spawnHeart, spawnChest, spawnPixelScene1, spawnOilTank, spawnPotion, spawnWand };

			for (int px = 0; px < map_w; px++)
			{
				for (int py = 0; py < map_h; py++)
				{
					int pixelPos = 3 * (px + py * map_w);

					int gpX = GetGlobalPosX(worldX, worldY, px * 10, py * 10);
					int gpY = GetGlobalPosY(worldX, worldY, px * 10, py * 10 - 40);

					int PWSize = (ngPlus > 0 ? 64 : 70) * 512;

					//avoids having to switch every loop
					auto func = spawnFuncs[0];
					long pix = createRGB(map[pixelPos], map[pixelPos + 1], map[pixelPos + 2]);

					switch (pix) {
					case 0x78ffff:
						func = spawnFuncs[0];
						break;
					case 0x55ff8c:
						func = spawnFuncs[1];
						break;
					case 0xff0aff:
						func = spawnFuncs[2];
						break;
					case 0xc35700:
						func = spawnFuncs[3];
						break;
					case 0x50a000:
						if (checkItems > 0)
							func = spawnFuncs[4];
						else continue;
						break;
					case 0x00ff00:
						if (checkWands > 0)
							func = spawnFuncs[5];
						else continue;
						break;
					default:
						continue;
					}

					for (int i = -pwCount; i <= pwCount; i++) {
						if (chestIdx >= (2 * pwCount + 1) * maxChestsPerWorld) {
							printf("Chest density exceeded in seed %i, with chest count above %i!\n", worldSeed, chestIdx);
							densityExceeded = true;
							break;
						}
						byte* c = retSegment + 4 + chestIdx * sizeOfChest();

						writeUnalignedInt(c, -1);
						writeUnalignedInt(c + 4, -1);
						*(c + 8) = 0;

						func(gpX + PWSize * i, gpY, worldSeed, greedCurse, expandSpells, c);

						if (readUnalignedInt(c) != -1) {
							if (loggingLevel >= 5) printf("Chest (%i %i) -> %i %i: %i\n", gpX, gpY, readUnalignedInt(c), readUnalignedInt(c + 4), *(c + 8));
							chestIdx++;
						}
					}
					if (densityExceeded) break;
				}
				if (densityExceeded) break;
			}
			writeUnalignedInt(retSegment, chestIdx);
		}
	}
}

__global__ void blockCheckEOE(
	int originX,
	int originY,
	uint radius,
	byte* retArray,
	byte checkItems,
	byte expandSpells,
	byte tinyMode) {
	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	uint stride = blockDim.x * gridDim.x;
	for (int idx = index; idx < radius * radius * 4; idx += stride) {
		byte* c = retArray + idx * sizeOfChest();
		memset(c, 0, sizeOfChest());

		int x = originX - radius + (idx % (radius * 2));
		int y = originY - radius +  (idx / (radius * 2));

		if (tinyMode > 0) {
			writeUnalignedInt(c, x);
			writeUnalignedInt(c + 4, y);
			*(c + 8) = 1;
			*(c + 9) = 64;
		}
		else
			CheckGreatChestLoot(x, y, worldSeedStart, c);

		if (loggingLevel >= 5) printf("Chest (%i %i) -> %i %i: %i\n", x, y, readUnalignedInt(c), readUnalignedInt(c + 4), *(c + 8));
	}
}

extern "C" {
#ifdef _MSC_VER
	__declspec(dllexport)
#else
	__attribute__((visibility("default")))
#endif
	byte** generate_block(
		byte host_tileData[],
		uint seeds[],
		uint tiles_w,
		uint tiles_h,
		uint _map_w,
		uint _map_h,
		bool _isCoalMine,
		byte _biomeIndex,
		int _worldX,
		int _worldY,
		uint _worldSeedStart,
		uint _worldSeedCount,
		uint maxTries,
		uint _pwCount,
		byte _ngPlus,
		byte _loggingLevel,
		uint _maxChestContents,
		uint _maxChestsPerWorld,
		byte _greedCurse,
		byte _checkItems,
		byte _expandSpells,
		byte _checkWands)
	{
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(map_w), &_map_w, sizeof(uint)));
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(map_h), &_map_h, sizeof(uint)));
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(isCoalMines), &_isCoalMine, sizeof(bool)));
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(biomeIndex), &_biomeIndex, sizeof(byte)));
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(worldX), &_worldX, sizeof(int)));
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(worldY), &_worldY, sizeof(int)));
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(worldSeedStart), &_worldSeedStart, sizeof(uint)));
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(worldSeedCount), &_worldSeedCount, sizeof(uint)));
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(pwCount), &_pwCount, sizeof(int)));
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(ngPlus), &_ngPlus, 1));
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(maxChestContents), &_maxChestContents, sizeof(uint)));
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(maxChestsPerWorld), &_maxChestsPerWorld, sizeof(uint)));
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(loggingLevel), &_loggingLevel, 1));

		byte* dTileData;
		checkCudaErrors(hipMalloc((void**)&dTileData, 3 * tiles_w * tiles_h));
		checkCudaErrors(hipMemcpy(dTileData, host_tileData, 3 * tiles_w * tiles_h, hipMemcpyHostToDevice));
		buildTS<<<1, 1>>>(dTileData, tiles_w, tiles_h);
		checkCudaErrors(hipDeviceSynchronize());
		checkCudaErrors(hipFree(dTileData));

		uint* dSeeds;
		byte* resultBlock = (byte*)malloc(3 * _map_w * _map_h * _worldSeedCount);
		byte* validBlock = (byte*)malloc(_worldSeedCount);

		NollaPrng* rngBlock1;
		NollaPrng* rngBlock2;
		byte* dResultBlock;
		byte* dResBlock;
		byte* dValidBlock;
		intPair* dStackMem;

		checkCudaErrors(hipMalloc((void**)&dSeeds, sizeof(uint) * _worldSeedCount));
		checkCudaErrors(hipMemcpy(dSeeds, seeds, sizeof(uint) * _worldSeedCount, hipMemcpyHostToDevice));

		checkCudaErrors(hipMalloc((void**)&rngBlock1, sizeof(NollaPrng) * _worldSeedCount));
		checkCudaErrors(hipMalloc((void**)&rngBlock2, sizeof(NollaPrng) * _worldSeedCount));
		checkCudaErrors(hipMalloc((void**)&dResultBlock, 3 * _map_w * _map_h * _worldSeedCount));
		checkCudaErrors(hipMalloc((void**)&dResBlock, 3 * _map_w * (_map_h + 4) * _worldSeedCount));
		checkCudaErrors(hipMalloc((void**)&dValidBlock, _worldSeedCount));
		checkCudaErrors(hipMalloc((void**)&dStackMem, sizeof(intPair) * _worldSeedCount * (_map_w + _map_h)));

		checkCudaErrors(hipMemset(dValidBlock, 0, _worldSeedCount));

		blockInitRNG<<<NUMBLOCKS, BLOCKSIZE>>>(dSeeds, rngBlock1, rngBlock2);
		checkCudaErrors(hipDeviceSynchronize());

		bool stop = false;

		int tries = 0;
		if (_loggingLevel >= 5) printf("Beginning generation attempts.\n");
		while (!stop) {
			//chrono::steady_clock::time_point time1 = chrono::steady_clock::now();
			if (tries >= maxTries) break;
			bool skipValid = tries == 0;

			blockUpdateRNG << <NUMBLOCKS, BLOCKSIZE >> > (rngBlock1, rngBlock2);
			checkCudaErrors(hipDeviceSynchronize());

			blockGenerateMap<<<NUMBLOCKS, BLOCKSIZE>>>(dResBlock, rngBlock2, dValidBlock, skipValid);
			checkCudaErrors(hipDeviceSynchronize());
			blockMemcpyOffset<<<NUMBLOCKS, BLOCKSIZE>>>(dResBlock, dResultBlock, dValidBlock, skipValid);
			checkCudaErrors(hipDeviceSynchronize());

			if (_isCoalMine) {
				blockCoalMineHax << <NUMBLOCKS, BLOCKSIZE >> > (dResultBlock, dValidBlock, skipValid);
				checkCudaErrors(hipDeviceSynchronize());
			}

			if (_worldY < 20 && _worldX > 32 && _worldX < 39) {
				blockRoomBlock<<<NUMBLOCKS, BLOCKSIZE>>>(dResultBlock, dValidBlock, skipValid);
				checkCudaErrors(hipDeviceSynchronize());
			}

			checkCudaErrors(hipMemset(dResBlock, 0, _worldSeedCount * _map_w * _map_h));
			blockIsValid<<<NUMBLOCKS, BLOCKSIZE>>>(dResultBlock, dValidBlock, dResBlock, dStackMem, skipValid);
			checkCudaErrors(hipDeviceSynchronize());

			checkCudaErrors(hipMemcpy(validBlock, dValidBlock, _worldSeedCount, hipMemcpyDeviceToHost));

			checkCudaErrors(hipMemcpy(resultBlock, dResultBlock, 3 * _map_w * _map_h * _worldSeedCount, hipMemcpyDeviceToHost));

			tries++;
			int numBad = 0;
			for (int j = 0; j < _worldSeedCount; j++) if (!validBlock[j]) { numBad++; }
			stop = numBad == 0;

			if(_loggingLevel >= 3) printf("Try %i: Maps invalid: %i\n", tries, numBad);
		}
		checkCudaErrors(hipFree(rngBlock1));
		checkCudaErrors(hipFree(rngBlock2));
		checkCudaErrors(hipFree(dResBlock));
		checkCudaErrors(hipFree(dStackMem));
		freeTS<<<1, 1>>>();
		checkCudaErrors(hipDeviceSynchronize());
		free(validBlock);

		byte* retArray = (byte*)malloc(_worldSeedCount * (sizeof(uint) + (9 + _maxChestContents) * _maxChestsPerWorld * (2 * _pwCount + 1)));
		byte* dRetArray;
		checkCudaErrors(hipMalloc((void**)&dRetArray, _worldSeedCount * (sizeof(uint) + (9 + _maxChestContents) * _maxChestsPerWorld * (2 * _pwCount + 1))));

		blockCheckSpawnables<<<NUMBLOCKS, BLOCKSIZE >>>(dSeeds, dResultBlock, dRetArray, dValidBlock, _greedCurse, _checkItems, _expandSpells, _checkWands);
		checkCudaErrors(hipDeviceSynchronize());

		checkCudaErrors(hipMemcpy(retArray, dRetArray, _worldSeedCount * (sizeof(uint) + (9 + _maxChestContents) * _maxChestsPerWorld * (2 * _pwCount + 1)), hipMemcpyDeviceToHost));

		checkCudaErrors(hipFree(dResultBlock));
		checkCudaErrors(hipFree(dValidBlock));
		checkCudaErrors(hipFree(dRetArray));

		byte** retList = (byte**)malloc(sizeof(byte*) * 2);
		retList[0] = retArray;
		retList[1] = resultBlock;
		return retList;
	}
}

extern "C" {
#ifdef _MSC_VER
	__declspec(dllexport)
#else
	__attribute__((visibility("default")))
#endif
		byte* search_eoe(
			int originX,
			int originY,
			uint radius,
			uint _worldSeed,
			byte _loggingLevel,
			uint _maxChestContents,
			byte checkItems,
			byte expandSpells,
			byte tinyMode
		) {
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(worldSeedStart), &_worldSeed, sizeof(uint)));
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(maxChestContents), &_maxChestContents, sizeof(uint)));
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(loggingLevel), &_loggingLevel, 1));

		byte* retArray = (byte*)malloc(((9 + _maxChestContents) * 4 * radius * radius));
		byte* dRetArray;
		checkCudaErrors(hipMalloc((void**)&dRetArray, (9 + _maxChestContents) * 4 * radius * radius));

		blockCheckEOE << <NUMBLOCKS, BLOCKSIZE >> > (originX, originY, radius, dRetArray, checkItems, expandSpells, tinyMode);
		checkCudaErrors(hipDeviceSynchronize());

		checkCudaErrors(hipMemcpy(retArray, dRetArray, (9 + _maxChestContents) * 4 * radius * radius, hipMemcpyDeviceToHost));
		checkCudaErrors(hipFree(dRetArray));

		return retArray;
	}
}


//I don't trust freeing memory in C#, better to just P/Invoke the pointer back to C++ and free it there
extern "C" {
#ifdef _MSC_VER
	__declspec(dllexport)
#else
	__attribute__((visibility("default")))
#endif
		void free_array(void* block) {
		free(block);
	}
}

int main() {
	return 0;
}